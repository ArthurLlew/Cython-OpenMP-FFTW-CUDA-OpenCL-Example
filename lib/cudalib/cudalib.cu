#include "hip/hip_runtime.h"
// Header file for this code
#include "cudalib.h"
// Complex numbers for device
#include <hip/hip_complex.h>


// Kernel code
__global__ void kernel(hipDoubleComplex *matrix, int n, int m)
{
    // i nad j are now obtained via blocks and threads
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    // Verify that we are not exceeding array bounds
    if (i < n && j < m)
    {
        matrix[i * m + j] = hipCadd(matrix[i * m + j], make_hipDoubleComplex(i, j));
    }
}


// Function to process FFT
void cuda_func(std::complex<double> *matrix, int n, int m)
{
    // Matrix size
    int size = sizeof(std::complex<double>)*n*m;
    // Make device memory pointer of a compatible complex type
    hipDoubleComplex *matrix_copy;
    // Threads and blocks settings
    // Since we know that we will get array 4x4, we can do block and thread management like this
    // In non-example code this small section should be more complicated
    dim3 threadsPerBlock(n/2, m/2);
    dim3 numBlocks(n/threadsPerBlock.x, m/threadsPerBlock.y);

    // Allocate memory
    hipMalloc(&matrix_copy, size);

    // Copy input to allocated memory
    hipMemcpy(matrix_copy, matrix, size, hipMemcpyHostToDevice);

    // Run kernel
    kernel<<<numBlocks, threadsPerBlock>>>(matrix_copy, n, m);

    // Retrieve result
    hipMemcpy(matrix, matrix_copy, size, hipMemcpyDeviceToHost);

    // Free allocated memory
    hipFree(matrix_copy);
}